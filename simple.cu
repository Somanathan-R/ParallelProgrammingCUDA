
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <tuple>

__device__ float deviceMultiply(float a, float b)
{
    return a * b;
}

__global__ void vectorMult(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = deviceMultiply(A[i], B[i]);
    }
}

__host__ std::tuple<float *, float *, float *> allocateHostMemory(int numElements)
{
    size_t size = numElements * sizeof(float);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    return std::make_tuple(h_A, h_B, h_C);
}

__host__ void executeKernel(float *d_A, float *d_B, float *d_C, int numElements)
{
    // Launch the Vector Multiply CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    vectorMult<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorMult kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main(void)
{
    int numElements = 50000;
    printf("[Vector multiplication of %d elements]\n", numElements);

    float *h_A, *h_B, *h_C;
    std::tie(h_A, h_B, h_C) = allocateHostMemory(numElements);
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, numElements * sizeof(float));
    hipMalloc(&d_B, numElements * sizeof(float));
    hipMalloc(&d_C, numElements * sizeof(float));

    // Copy inputs to device
    hipMemcpy(d_A, h_A, numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, numElements * sizeof(float), hipMemcpyHostToDevice);

    // Execute the kernel
    executeKernel(d_A, d_B, d_C, numElements);

    // Copy result back to host
    hipMemcpy(h_C, d_C, numElements * sizeof(float), hipMemcpyDeviceToHost);

    // Verify results
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs((h_A[i] * h_B[i]) - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}
